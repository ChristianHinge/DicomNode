#ifndef PLAYGROUND_CONSTANTS_H
#define PLAYGROUND_CONSTANTS_H

#include <stdio.h>

#define BLOCKSIZE 32

#define CUDA_RT_CALL(call)                                                                  \
    {                                                                                       \
        hipError_t cudaStatus = call;                                                      \
        if (hipSuccess != cudaStatus) {                                                    \
            fprintf(stderr,                                                                 \
                    "ERROR: CUDA RT call \"%s\" in line %d of file %s failed "              \
                    "with "                                                                 \
                    "%s (%d).\n",                                                           \
                    #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus); \
            exit(cudaStatus);                                                               \
        }                                                                                   \
  }

typedef std::pair<int,int> index2;

struct Index {
  int y;
  int x;
};


#endif